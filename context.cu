#include "context.cuh"
#include <iostream>
#include "check.hpp"

bool is_context = false;
hipCtx_t context;

hipCtx_t get_context()
{
	if (!is_context)
	{
		check(hipInit(0));
		hipDevice_t device;
		check(hipDeviceGet(&device, 0));
		check(hipCtxCreate(&context, 0, device));
		is_context = true;
	}
	return context;
}



